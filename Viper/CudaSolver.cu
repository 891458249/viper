#include "hip/hip_runtime.h"
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     https://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "CollisionGrid.cuh"
#include "ConstraintsInfo.h"
#include "CudaConstraints.cuh"
#include "CudaData.cuh"
#include "CudaSolver.h"
#include "CudaUtils.cuh"
#include <fstream>
#include <thrust/binary_search.h>
#include <thrust/gather.h>

#define MIN_RADIUS 0.001f
#define COLLISION_MARGIN 0.002f

namespace viper {

struct CudaConstraints {
    thrust::device_vector<C_skinning> skinning;
    thrust::device_vector<C_distance> dist;
    thrust::device_vector<C_distancemax> distmax;
    thrust::device_vector<C_volume> vol;
    thrust::device_vector<C_volume2> vol2;
    thrust::device_vector<C_bend> bend;
    thrust::device_vector<C_stretch> stretch;
    thrust::device_vector<C_radius> radius;
    thrust::device_vector<C_shape> shape;
    thrust::device_vector<C_shape2> shape2;
    thrust::device_vector<C_touch> touch;
    thrust::device_vector<C_bilap> bilap;
    thrust::device_vector<C_collpp> collpp;
    thrust::device_vector<C_collision> collision;
};

struct CudaSolverData {
    CudaSimData S;      // state
    CudaProjections Pc; // projections per constraint
    CudaProjections Pp; // projections per particle
    CudaProjections Pt; // projections per particle temp
    CudaConstraints C;  // constraints
    CollisionGrid cgrid;

    thrust::device_vector<int> c_perm;
};

struct floor_friction {
    CudaStatePtr state;

    floor_friction(CudaStatePtr S) : state(S) {}

    __device__ void operator()(int i) const {
        if (state.xa[i] == 0)
            return;

        bool isTouching = state.x[i][1] - state.r[i] < 1e-6f;
        if (state.w[i] > 1e-6f && isTouching) {
            float pen = min(0.f, state.x[i][1] - state.r[i]);
            Vec3 dx = state.x[i] - state.xp[i];
            Vec3 tandx = Vec3(dx[0], 0.f, dx[2]);
            float tan_norm = tandx.norm();
            float mu_s = 0.01;
            float mu_k = 3.0;
            float factor = 0.99f;

            float d = abs(pen);
            if (tan_norm > mu_s * d)
                factor = min(0.99, mu_k * d / tan_norm);

            state.x[i][1] -= pen; // normal y
            state.x[i][0] -= tandx[0] * factor; // tangential x
            state.x[i][2] -= tandx[2] * factor; // tangential z
        }
        float wall = 20.0f;
        float newx = max(-wall, min(wall, state.x[i][0]));
        float newz = max(-wall, min(wall, state.x[i][2]));

    }
};

struct V_integration {
    CudaStatePtr state;
    float dt;
    Vec3 gravity;
    float damping;

    V_integration(CudaStatePtr S, float dt, Vec3 g, float damping)
        : state(S), dt(dt), gravity(g), damping(damping) {}

    __device__ void operator()(int i) const {
        if (state.xa[i] == 0)
            return;

        Vec3 v = Vec3::Zero();
        if (state.w[i] > 1e-6f)
            v = ((state.x[i] - state.xp[i]) / dt + dt * gravity) * damping;
        state.xp[i] = state.x[i];
        state.x[i] += dt * v;

        float dr = 0.f;
        if (state.wr[i] > 1e-6f)
            dr = (state.r[i] - state.rp[i]) * damping;
        state.rp[i] = state.r[i];
        state.r[i] += dr;
    }
};

struct Vq_integration {
    CudaStatePtr state;
    float dt;
    float damping;

    Vq_integration(CudaStatePtr S, float dt, float damping)
        : state(S), dt(dt), damping(damping) {}

    __device__ void operator()(int i) const {
        if (state.qa[i] == 0)
            return;

        Vec3 vq = 2.f / dt * (state.qp[i].conjugate() * state.q[i]).vec();
        vq *= damping;
        Quaternion vqq;
        vqq.w() = 0.f;
        vqq.vec() = vq;
        state.qp[i] = state.q[i];
        state.q[i] =
            state.qp[i].coeffs() + 0.5f * dt * (state.qp[i] * vqq).coeffs();
        state.q[i].normalize();
    }
};

struct bend_damping {
    CudaStatePtr state;
    C_bend *C;
    float dt;
    float damping;

    __device__ void operator()(int i) const {
        Quaternion &qa = state.q[C[i].a];
        Quaternion &qb = state.q[C[i].b];
        Quaternion &qap = state.qp[C[i].a];
        Quaternion &qbp = state.qp[C[i].b];

        Vec3 vqa = 2.f / dt * (qap.conjugate() * qa).vec();
        Vec3 vqb = 2.f / dt * (qbp.conjugate() * qb).vec();

        Vec3 dv = (vqb - vqa) * (1.0f - damping);

        vqa += dv;
        vqb -= dv;

        Quaternion vqaq, vqbq;
        vqaq.w() = 0.f;
        vqbq.w() = 0.f;

        vqaq.vec() = vqa;
        vqbq.vec() = vqb;

        qa = qap.coeffs() + 0.5f * dt * (qap * vqaq).coeffs();
        qb = qbp.coeffs() + 0.5f * dt * (qbp * vqbq).coeffs();

        qa.normalize();
        qb.normalize();
    }
};

struct apply_projection_particles {
    Vec3 *x;
    float *r;
    Vec6 *dx;
    int *id;
    uint8_t *a;

    apply_projection_particles(Vec3 *x, float *r, Vec6 *dx, int *id, uint8_t *a)
        : x(x), r(r), dx(dx), id(id), a(a) {}

    __device__ void operator()(int i) const {
        int k = id[i];
        if (a[k] == 0)
            return;

        if (dx[i][4] > 1e-6f)
            x[k] += dx[i].head<3>() / dx[i][4];

        if (dx[i][5] > 1e-6f)
            r[k] = fmaxf(MIN_RADIUS, r[k] + dx[i][3] / dx[i][5]);
    }
};

struct apply_projection_frames {
    Quaternion *x;
    Vec6 *dx;
    int *id;
    int N;
    uint8_t *a;

    apply_projection_frames(Quaternion *x, Vec6 *dx, int *id, uint8_t *a, int N)
        : x(x), dx(dx), id(id), N(N), a(a) {}

    __device__ void operator()(int i) const {
        int k = id[i] - N;
        if (a[k] == 0)
            return;

        if (dx[i][4] > 1e-6f)
            x[k].coeffs() += dx[i].head<4>() / dx[i][4];
        x[k].normalize();
    }
};

struct generate_pills_proxys {
    Vec3 *x;
    float *r;
    Vec2i *pills;
    Vec3 *sx;
    float *sr;

    generate_pills_proxys(Vec3 *x, float *r, Vec2i *pills, Vec3 *sx, float *sr)
        : x(x), r(r), pills(pills), sx(sx), sr(sr) {}

    __device__ void operator()(int i) const {
        int a = pills[i][0];
        int b = pills[i][1];
        Vec3 s0 = x[a];
        Vec3 s1 = x[b];
        float r0 = r[a];
        float r1 = r[b];
        Vec3 d = s1 - s0;
        float l = d.norm();
        Vec3 dl = d / (l + FLT_EPSILON);
        sx[i] = (s1 + s0 + dl * (r1 - r0)) / 2;
        sr[i] = (l + r0 + r1) / 2;
    }
};

struct generate_collisions {
    Vec2i *pills;
    const Vec2i *coll_pairs;
    C_collision *C;

    generate_collisions(Vec2i *pills, const Vec2i *coll_pairs, C_collision *C)
        : pills(pills), coll_pairs(coll_pairs), C(C) {}

    __device__ void operator()(int i) const {
        int a = coll_pairs[i][0];
        int b = coll_pairs[i][1];
        C[i].a = pills[a];
        C[i].b = pills[b];
        C[i].enabled = true;
    }
};

struct collision_filter {
    Vec2i *pills;
    int *group;
    CudaStatePtr S;

    collision_filter(Vec2i *pills, int *group, CudaStatePtr S)
        : pills(pills), group(group), S(S) {}

    __device__ bool operator()(const Vec2i &c) {
        int a0 = pills[c[0]][0];
        int a1 = pills[c[0]][1];
        int b0 = pills[c[1]][0];
        int b1 = pills[c[1]][1];
        int zeroa = S.w[a0] < 1e-6f || S.w[a1] < 1e-6f;
        int zerob = S.w[b0] < 1e-6f || S.w[b1] < 1e-6f;

        return group[c[0]] != group[c[1]] && (zeroa + zerob < 2);
    }
};

CudaSolver::CudaSolver() { gpu = new CudaSolverData(); }

CudaSolver::~CudaSolver() {
    // delete gpu;
}

template <typename T> struct DisabledPredicate {
    bool operator()(const T &constraint) { return !constraint.enabled; }
};

template <typename CPUVec, typename GPUVec>
void upload_and_filter(GPUVec &gpu_vec, const CPUVec &cpu_vec) {
    using T = typename GPUVec::value_type;
    gpu_vec = cpu_vec;
    if (gpu_vec.size() > 0) {
        gpu_vec.erase(thrust::remove_if(thrust::device, gpu_vec.begin(),
                                        gpu_vec.end(), DisabledPredicate<T>()),
                      gpu_vec.end());
    }
}

double CudaSolver::solve(ConstraintsCPU &constraints, SimulationState &state,
                         const std::vector<Vec2i> &pills,
                         const std::vector<int> &group, float dt, const Vec3 &g,
                         int iterations, bool floor, float damping) {
    CudaConstraints &C = gpu->C;
    CudaSimData &S = gpu->S;
    CudaProjections &Pc = gpu->Pc;
    CudaProjections &Pt = gpu->Pt;
    CudaProjections &Pp = gpu->Pp;

    // CPU -> GPU
    S.X.x = state.x;
    S.X.q = state.q;
    S.X.r = state.r;

    S.Xp.x = state.xp;
    S.Xp.q = state.qp;
    S.Xp.r = state.rp;

    S.Xi.x = state.xi;
    S.Xi.q = state.qi;
    S.Xi.r = state.ri;

    S.b = state.b;
    S.bp = state.bp;
    S.bi = state.bi;

    S.w = state.w;
    S.wq = state.wq;
    S.wr = state.wr;

    S.xa = state.xa;
    S.qa = state.qa;

    thrust::device_vector<Vec2i> gpu_pills = pills;
    thrust::device_vector<int> pill_groups = group;

    int N = state.x.size(); // particles count
    int M = state.q.size(); // pills count

    upload_and_filter(C.dist, constraints.distance);
    upload_and_filter(C.distmax, constraints.distancemax);
    upload_and_filter(C.skinning, constraints.skinning);
    upload_and_filter(C.vol, constraints.volume);
    upload_and_filter(C.vol2, constraints.volume2);
    upload_and_filter(C.bend, constraints.bend);
    upload_and_filter(C.stretch, constraints.stretch);
    upload_and_filter(C.bilap, constraints.bilap);
    upload_and_filter(C.shape, constraints.shape);
    upload_and_filter(C.shape2, constraints.shape2);
    upload_and_filter(C.radius, constraints.radius);
    upload_and_filter(C.touch, constraints.touch);

    tic();

    // time integration
    thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(N),
                     V_integration(CudaStatePtr(S), dt, g, damping));
    thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(M),
                     Vq_integration(CudaStatePtr(S), dt, damping));
    thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator((int)C.skinning.size()),
                     C_skinning_solve(ptr(C.skinning), CudaStatePtr(S)));

    float t_velocity = toc();
    thrust::device_vector<Vec3> sp(M);
    thrust::device_vector<float> sr(M);
    thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(M),
                     generate_pills_proxys(ptr(S.X.x), ptr(S.X.r),
                                           ptr(gpu_pills), ptr(sp), ptr(sr)));
    if (M < 2) {
        C.collision.resize(0);
    } else {
        const thrust::device_vector<Vec2i> &coll_pairs =
            gpu->cgrid.test_particles(sp, sr, COLLISION_MARGIN);
        thrust::device_vector<Vec2i> coll_pairs_filtered(coll_pairs.size());
        auto valid_coll_end =
            thrust::copy_if(thrust::device, coll_pairs.begin(),
                            coll_pairs.end(), coll_pairs_filtered.begin(),
                            collision_filter(ptr(gpu_pills), ptr(pill_groups),
                                             CudaStatePtr(S)));
        coll_pairs_filtered.erase(valid_coll_end, coll_pairs_filtered.end());
        int K = coll_pairs_filtered.size();
        C.collision.resize(K);
        thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(K),
                         generate_collisions(ptr(gpu_pills),
                                             ptr(coll_pairs_filtered),
                                             ptr(C.collision)));
    }

    float t_collision = toc();
    ConstraintsInfo cInfo;
    cInfo.add("distance", C.dist.size(), 2, 1);
    cInfo.add("volume", C.vol.size(), 2, 1);
    cInfo.add("volume2", C.vol2.size(), 3, 1);
    cInfo.add("bend", C.bend.size(), 2, 3);
    cInfo.add("stretch", C.stretch.size(), 3, 3);
    cInfo.add("bilap", C.bilap.size(), 1, 1);
    cInfo.add("shape", C.shape.size(), SHAPE_MATCHING_MAX, 1);
    cInfo.add("shape2", C.shape2.size(), 3 * SHAPE_MATCHING_MAX, 1);
    cInfo.add("radius", C.radius.size(), 1, 1);
    cInfo.add("collision", C.collision.size(), 4, 0);
    cInfo.add("touch", C.touch.size(), 2, 0);

    int np = cInfo.get_np();
    int nl = cInfo.get_nl();
    std::map<std::string, int> o = cInfo.get_o();
    std::map<std::string, int> ol = cInfo.get_ol();

    int n_cst = C.dist.size() + C.vol.size() + C.bend.size() +
                C.stretch.size() + C.shape.size() + C.radius.size() +
                C.collision.size() + C.vol2.size() + C.shape2.size();
    bool permutation_built = false;
    Pc.resize(np);
    Pt.resize(np);
    Pp.resize(N + M);
    thrust::device_vector<float> L(nl); // XPBD
    thrust::fill(L.begin(), L.end(), 0.f);
    thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator((int)C.bend.size()),
                     bend_damping{CudaStatePtr(S), ptr(C.bend), dt, 0.98f});

    for (int i = 0; i < iterations; i++) {
        bool collisions_only = (i % 2 == 1) || i == -1;
        Pc.setZero();
        Pp.setZero();
        if (!collisions_only) {
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.dist.size()),
                C_distance_solve(ptr(C.dist), CudaStatePtr(S),
                                 CudaProjectionsPtr(Pc, o["distance"]),
                                 ptr(L, ol["distance"]), dt));
            thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator((int)C.vol.size()),
                             C_volume_solve(ptr(C.vol), CudaStatePtr(S),
                                            CudaProjectionsPtr(Pc, o["volume"]),
                                            ptr(L, ol["volume"]), dt));
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.vol2.size()),
                C_volume2_solve(ptr(C.vol2), CudaStatePtr(S),
                                CudaProjectionsPtr(Pc, o["volume2"]),
                                ptr(L, ol["volume2"]), dt));
            thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator((int)C.bend.size()),
                             C_bend_solve(ptr(C.bend), CudaStatePtr(S),
                                          CudaProjectionsPtr(Pc, o["bend"]), N,
                                          ptr(L, ol["bend"]), dt));
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.stretch.size()),
                C_stretch_solve(ptr(C.stretch), CudaStatePtr(S),
                                CudaProjectionsPtr(Pc, o["stretch"]), N,
                                ptr(L, ol["stretch"]), dt));
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.bilap.size()),
                C_bilap_solve(ptr(C.bilap), CudaStatePtr(S),
                              CudaProjectionsPtr(Pc, o["bilap"]),
                              ptr(L, ol["bilap"]), dt));
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.shape.size()),
                C_shape_solve(ptr(C.shape), CudaStatePtr(S),
                              CudaProjectionsPtr(Pc, o["shape"]),
                              ptr(L, ol["shape"]), dt));
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.shape2.size()),
                C_shape2_solve(ptr(C.shape2), CudaStatePtr(S),
                               CudaProjectionsPtr(Pc, o["shape2"]), N,
                               ptr(L, ol["shape2"]), dt));
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.radius.size()),
                C_radius_solve(ptr(C.radius), CudaStatePtr(S),
                               CudaProjectionsPtr(Pc, o["radius"]),
                               ptr(L, ol["radius"]), dt));
            thrust::for_each(
                thrust::device, thrust::make_counting_iterator(0),
                thrust::make_counting_iterator((int)C.touch.size()),
                C_touch_solve(ptr(C.touch), CudaStatePtr(S),
                              CudaProjectionsPtr(Pc, o["touch"]),
                              ptr(L, ol["touch"]), dt));
        }
        thrust::for_each(
            thrust::device, thrust::make_counting_iterator(0),
            thrust::make_counting_iterator((int)C.collision.size()),
            C_collision_solve(ptr(C.collision), CudaStatePtr(S),
                              CudaProjectionsPtr(Pc, o["collision"])));

        if (!permutation_built) {
            gpu->c_perm.resize(np);
            thrust::sequence(thrust::device, gpu->c_perm.begin(),
                             gpu->c_perm.end());
            auto vals_begin = thrust::make_zip_iterator(
                thrust::make_tuple(Pc.dx.begin(), gpu->c_perm.begin()));
            thrust::sort_by_key(Pc.id.begin(), Pc.id.end(), vals_begin);
            Pt = Pc;
            permutation_built = true;
        } else {
            auto src_begin = thrust::make_zip_iterator(
                thrust::make_tuple(Pc.dx.begin(), Pc.id.begin()));
            auto dst_begin = thrust::make_zip_iterator(
                thrust::make_tuple(Pt.dx.begin(), Pt.id.begin()));
            thrust::gather(thrust::device, gpu->c_perm.begin(),
                           gpu->c_perm.end(), src_begin, dst_begin);
        }
        auto new_end =
            thrust::reduce_by_key(thrust::device, Pt.id.begin(), Pt.id.end(),
                                  Pt.dx.begin(), Pp.id.begin(), Pp.dx.begin())
                .first;
        auto f_start = thrust::lower_bound(Pp.id.begin(), new_end, N);
        int proj_count = new_end - Pp.id.begin();
        int p_count = f_start - Pp.id.begin();
        thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(p_count),
                         apply_projection_particles(ptr(S.X.x), ptr(S.X.r),
                                                    ptr(Pp.dx), ptr(Pp.id),
                                                    ptr(S.xa)));
        thrust::for_each(thrust::device,
                         thrust::make_counting_iterator(p_count),
                         thrust::make_counting_iterator(proj_count),
                         apply_projection_frames(ptr(S.X.q), ptr(Pp.dx),
                                                 ptr(Pp.id), ptr(S.qa), N));

        if (floor)
            thrust::for_each(thrust::device, thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator(N),
                             floor_friction(CudaStatePtr(S)));
    }
    float t_solve = toc();

    // GPU -> CPU
    thrust::copy(S.X.x.begin(), S.X.x.end(), state.x.begin());
    thrust::copy(S.Xp.x.begin(), S.Xp.x.end(), state.xp.begin());
    thrust::copy(S.X.q.begin(), S.X.q.end(), state.q.begin());
    thrust::copy(S.Xp.q.begin(), S.Xp.q.end(), state.qp.begin());
    thrust::copy(S.X.r.begin(), S.X.r.end(), state.r.begin());
    thrust::copy(S.Xp.r.begin(), S.Xp.r.end(), state.rp.begin());
    thrust::copy(C.shape.begin(), C.shape.end(), constraints.shape.begin());
    thrust::copy(C.shape2.begin(), C.shape2.end(), constraints.shape2.begin());

    return t_solve;
}

} // namespace viper