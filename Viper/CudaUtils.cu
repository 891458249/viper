// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     https://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "CudaUtils.cuh"
#include <chrono>

namespace viper {

decltype(std::chrono::system_clock::now()) start_time;

void tic() {
    hipDeviceSynchronize();
    start_time = std::chrono::system_clock::now();
}

double toc() {
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();

    auto dur = end - start_time;
    auto us =
        std::chrono::duration_cast<std::chrono::microseconds>(dur).count();

    return (double)us / 1000.0;
}

} // namespace viper