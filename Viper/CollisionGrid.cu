#include "hip/hip_runtime.h"
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     https://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "CollisionGrid.cuh"

#include <chrono>
#include <iomanip>

#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/iterator/counting_iterator.h>

#include "CudaUtils.cuh"

#define SK_DEVICE __host__ __device__

namespace viper {

Vec4 CollisionGrid::b_sphere;

const thrust::device_vector<Vec2i> &
CollisionGrid::test_particles(const thrust::device_vector<Vec3> &c_in,
                              const thrust::device_vector<float> &r_in,
                              float eps) {

    float total_ms = 0;
    float time_ms = 0;

    auto zero_begin = thrust::make_counting_iterator(0);

    int n_particles = c_in.size();

    // Input particle positions
    c = c_in;
    // Input particles radii
    r.resize(r_in.size());
    auto add_margin = [=] SK_DEVICE(float x) -> float {
        return x + 0.5f * eps;
    };
    thrust::transform(thrust::device, r_in.begin(), r_in.end(), r.begin(),
                      add_margin);

    // Particle IDs
    i.clear();
    i.insert(i.end(), zero_begin, zero_begin + n_particles);

    int max_i =
        thrust::max_element(thrust::device, r.begin(), r.end()) - r.begin();

    float max_radius = r_in[max_i];

    auto part_to_sphere =
        [=] SK_DEVICE(thrust::tuple<Vec3, float, int> part) -> Vec4 {
        return Vec4(part.get<0>()[0], part.get<0>()[1], part.get<0>()[2],
                    part.get<1>());
    };

    auto particles_begin = thrust::make_zip_iterator(
        thrust::make_tuple(c.begin(), r.begin(), i.begin()));

    auto spheres_begin =
        thrust::make_transform_iterator(particles_begin, part_to_sphere);

    using Vec6 = Eigen::Matrix<float, 6, 1>;

    Vec3 c_init = c_in[0];
    float r_init = r_in[0];

    auto sphere_bbox = [=] SK_DEVICE(Vec4 s) -> Vec6 {
        Vec6 bbox;
        bbox.segment<3>(0) = s.head<3>() - Vec3::Ones() * s[3];
        bbox.segment<3>(3) = s.head<3>() + Vec3::Ones() * s[3];
        return bbox;
    };

    auto bbox_union = [=] SK_DEVICE(Vec6 b0, Vec6 b1) -> Vec6 {
        Vec6 result;
        result.segment<3>(0) = b0.segment<3>(0).cwiseMin(b1.segment<3>(0));
        result.segment<3>(3) = b0.segment<3>(3).cwiseMax(b1.segment<3>(3));
        return result;
    };

    auto bboxes_begin =
        thrust::make_transform_iterator(spheres_begin, sphere_bbox);

    Vec6 bbox_init = sphere_bbox(Vec4(c_init[0], c_init[1], c_init[2], r_init));

    Vec6 bbox =
        thrust::reduce(thrust::device, bboxes_begin, bboxes_begin + n_particles,
                       bbox_init, bbox_union);

    Vec3 bbox_min = bbox.segment<3>(0);
    Vec3 bbox_max = bbox.segment<3>(3);

    b_sphere.head<3>() = (bbox_max + bbox_min) / 2;
    b_sphere[3] = (bbox_max - bbox_min).norm() / 2;

    float grid_width = bbox_max[0] - bbox_min[0];
    int grid_res = grid_width / (2 * max_radius);
    grid_res = max(min(grid_res, 32), 1);

    int n_cells = grid_res * grid_res * grid_res;

    float cell_size = grid_width / grid_res;

    auto center_to_cell_id = [=] SK_DEVICE(Vec3 c) {
        Vec3i i = ((c - bbox_min) / cell_size)
                      .cast<int>()
                      .cwiseMax(Vec3i::Zero())
                      .cwiseMin(Vec3i::Ones() * (grid_res - 1));
        return i[0] + grid_res * i[1] + grid_res * grid_res * i[2];
    };

    auto part_cell_ids_gen =
        thrust::make_transform_iterator(c.begin(), center_to_cell_id);

    // ID of cell containing each particle
    part_cell_ids.clear();
    part_cell_ids.insert(part_cell_ids.end(), part_cell_ids_gen,
                         part_cell_ids_gen + n_particles);

    // Sort particle list by id of cells
    thrust::sort_by_key(thrust::device, part_cell_ids.begin(),
                        part_cell_ids.end(), particles_begin);

    // Indices of first particle for each cell
    cell_starts.clear();
    cell_starts.resize(n_cells, 0);
    cell_ends.clear();
    cell_ends.resize(n_cells, 0);

    auto part_cell_ids_begin = thrust::raw_pointer_cast(part_cell_ids.data());
    auto cell_starts_begin = thrust::raw_pointer_cast(cell_starts.data());
    auto cell_ends_begin = thrust::raw_pointer_cast(cell_ends.data());

    auto write_cell_starts = [=] SK_DEVICE(int i) {
        int id1 = part_cell_ids_begin[i];

        if (i == 0) {
            cell_starts_begin[id1] = i;
        } else {
            int id0 = part_cell_ids_begin[i - 1];
            if (id0 != id1) {
                cell_starts_begin[id1] = i;
                cell_ends_begin[id0] = i;
            }
        }
        if (i == n_particles - 1) {
            cell_ends_begin[id1] = n_particles;
        }
    };

    // Write out start indices of each cell in sorted particle list
    thrust::for_each(thrust::device, zero_begin, zero_begin + n_particles,
                     write_cell_starts);

    // Number of particles in each cell
    parts_per_cell.clear();
    parts_per_cell.resize(n_cells, 0);

    auto count_cell_parts = [=] SK_DEVICE(int cell) {
        return cell_ends_begin[cell] - cell_starts_begin[cell];
    };

    thrust::transform(thrust::device, zero_begin, zero_begin + n_cells,
                      parts_per_cell.begin(), count_cell_parts);

    auto parts_per_cell_begin = thrust::raw_pointer_cast(parts_per_cell.data());

    auto n_neighbours = [=] SK_DEVICE(int pair_id) {
        int nhbr_off = pair_id % 27;
        int particle_id = pair_id / 27;
        int cell_id = part_cell_ids_begin[particle_id];

        int c_i = cell_id % grid_res;
        int c_j = (cell_id / grid_res) % grid_res;
        int c_k = cell_id / (grid_res * grid_res);

        int i = (nhbr_off % 3) - 1;
        int j = ((nhbr_off / 3) % 3) - 1;
        int k = (nhbr_off / 9) - 1;

        int n_i = c_i + i;
        int n_j = c_j + j;
        int n_k = c_k + k;

        int nhbr_cell_id = n_i + grid_res * n_j + grid_res * grid_res * n_k;

        bool inside = n_i >= 0 && n_i < grid_res && n_j >= 0 &&
                      n_j < grid_res && n_k >= 0 && n_k < grid_res;

        int count = 0;
        if (inside) {
            count = parts_per_cell_begin[nhbr_cell_id];
        }

        return count;
    };

    auto neighbour_count_begin =
        thrust::make_transform_iterator(zero_begin, n_neighbours);

    int n_pairs = n_particles * 27;

    // Count potential collisions for each particle-cell pair
    // NOTE: promote from 27 threads per particle to 32 for warp coherence if
    // slow
    neighbours_per_pair.clear();
    neighbours_per_pair.insert(neighbours_per_pair.end(), neighbour_count_begin,
                               neighbour_count_begin + n_pairs);

    // Index of first collision for each particle-cell pair
    pair_collision_starts.resize(n_pairs);

    // tic();

    // Find start indices for cell-particle_pairs
    thrust::exclusive_scan(thrust::device, neighbours_per_pair.begin(),
                           neighbours_per_pair.end(),
                           pair_collision_starts.begin());

    int n_collisions =
        pair_collision_starts.back() + neighbours_per_pair.back();

    // Number of particle-cell pairs pointing to a group of collisions
    pairs_per_coll_group.clear();
    pairs_per_coll_group.resize(n_pairs, 1);

    // In-place scan to compute above count
    thrust::inclusive_scan_by_key(thrust::device, pair_collision_starts.begin(),
                                  pair_collision_starts.end(),
                                  pairs_per_coll_group.begin(),
                                  pairs_per_coll_group.begin());

    // Compute vector with number of pairs pointing to the start of each
    // collision group
    collision_pair_ids.clear();
    collision_pair_ids.resize(n_collisions, 0);
    thrust::scatter_if(thrust::device, pairs_per_coll_group.begin(),
                       pairs_per_coll_group.end(),
                       pair_collision_starts.begin(),
                       neighbours_per_pair.begin(), collision_pair_ids.begin());

    // Compute pair id of each collision
    thrust::inclusive_scan(thrust::device, collision_pair_ids.begin(),
                           collision_pair_ids.end(),
                           collision_pair_ids.begin());

    // Subtract one to get indices
    thrust::for_each(thrust::device, collision_pair_ids.begin(),
                     collision_pair_ids.end(),
                     [] SK_DEVICE(int &i) { i -= 1; });

    auto collision_pair_ids_begin =
        thrust::raw_pointer_cast(collision_pair_ids.data());
    auto pair_collision_starts_begin =
        thrust::raw_pointer_cast(pair_collision_starts.data());

    auto collision_generator = [=] SK_DEVICE(int collision_id) {
        int pair_id = collision_pair_ids_begin[collision_id];
        int collision_start = pair_collision_starts_begin[pair_id];
        int part_offset = collision_id - collision_start;

        int nhbr_off = pair_id % 27;
        int particle_id = pair_id / 27;
        int cell_id = part_cell_ids_begin[particle_id];

        int c_i = cell_id % grid_res;
        int c_j = (cell_id / grid_res) % grid_res;
        int c_k = cell_id / (grid_res * grid_res);

        int i = (nhbr_off % 3) - 1;
        int j = ((nhbr_off / 3) % 3) - 1;
        int k = (nhbr_off / 9) - 1;

        int n_i = c_i + i;
        int n_j = c_j + j;
        int n_k = c_k + k;

        int nhbr_cell_id = n_i + grid_res * n_j + grid_res * grid_res * n_k;

        int cell_start = cell_starts_begin[nhbr_cell_id];

        int other_particle_id = cell_start + part_offset;

        return Vec2i(particle_id, other_particle_id);
    };

    auto collisions_gen =
        thrust::make_transform_iterator(zero_begin, collision_generator);

    // Generate particle ids for all potential collisions
    collisions.clear();
    collisions.insert(collisions.end(), collisions_gen,
                      collisions_gen + n_collisions);

    auto c_begin = thrust::raw_pointer_cast(c.data());
    auto r_begin = thrust::raw_pointer_cast(r.data());

    auto not_colliding = [=] SK_DEVICE(Vec2i collision) {
        Vec3 c0 = c_begin[collision[0]];
        Vec3 c1 = c_begin[collision[1]];
        float r0 = r_begin[collision[0]];
        float r1 = r_begin[collision[1]];

        return (c1 - c0).norm() > (r0 + r1) || (collision[0] >= collision[1]);
    };

    auto valid_coll_end = thrust::remove_if(thrust::device, collisions.begin(),
                                            collisions.end(), not_colliding);

    collisions.erase(valid_coll_end, collisions.end());

    auto i_begin = thrust::raw_pointer_cast(i.data());

    auto permute_indices = [=] SK_DEVICE(Vec2i & coll) {
        coll[0] = i_begin[coll[0]];
        coll[1] = i_begin[coll[1]];
    };

    thrust::for_each(thrust::device, collisions.begin(), collisions.end(),
                     permute_indices);

    return collisions;
}

} // namespace viper